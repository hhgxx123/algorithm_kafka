#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>

#include "Common.h"
namespace bdavs {
__global__ void gpuPreImageScaleMean(unsigned char* input, int inputWidth, int inputHeight, int inputChannels,
                                   float* output, int outputWidth, int outputHeight,
                                   float3 scale, const float3 mean_value,const int color_type)
{
    //2D Index of current thread
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    if(dx < outputWidth && dy < outputHeight)
    {
        // BGRA or BGR
        if(inputChannels == 4 || inputChannels == 3)
        {

            double scale_x = (double) inputWidth / outputWidth;
            double scale_y = (double) inputHeight / outputHeight;

            int xmax = outputWidth;

            float fx = (float)((dx + 0.5) * scale_x - 0.5);
            int sx = floor(fx);
            fx = fx - sx;

            int isx1 = sx;
            if (isx1 < 0)
            {
                fx = 0.0;
                isx1 = 0;
            }
            if (isx1 >= (inputWidth - 1))
            {
                xmax = ::min( xmax, dy);
                fx = 0;
                isx1 = inputWidth - 1;
            }

            float2 cbufx;
            cbufx.x = (1.f - fx);
            cbufx.y = fx;

            float fy = (float)((dy + 0.5) * scale_y - 0.5);
            int sy = floor(fy);
            fy = fy - sy;

            int isy1 = clip(sy - 1 + 1 + 0, 0, inputHeight);
            int isy2 = clip(sy - 1 + 1 + 1, 0, inputHeight);

            float2 cbufy;
            cbufy.x = (1.f - fy);
            cbufy.y = fy;

            int isx2 = isx1 + 1;

            float3 d0;

            float3 s11 = make_float3(input[(isy1 * inputWidth + isx1) * inputChannels + 0] ,
                    input[(isy1 * inputWidth + isx1) * inputChannels + 1] ,
                    input[(isy1 * inputWidth + isx1) * inputChannels + 2]);

            float3 s12 = make_float3(input[(isy1 * inputWidth + isx2) * inputChannels + 0] ,
                    input[(isy1 * inputWidth + isx2) * inputChannels + 1] ,
                    input[(isy1 * inputWidth + isx2) * inputChannels + 2]);

            float3 s21 = make_float3(input[(isy2 * inputWidth + isx1) * inputChannels + 0] ,
                    input[(isy2 * inputWidth + isx1) * inputChannels + 1] ,
                    input[(isy2 * inputWidth + isx1) * inputChannels + 2]);

            float3 s22 = make_float3(input[(isy2 * inputWidth + isx2) * inputChannels + 0] ,
                    input[(isy2 * inputWidth + isx2) * inputChannels + 1] ,
                    input[(isy2 * inputWidth + isx2) * inputChannels + 2]);

            float h_rst00, h_rst01;
            // B
            if( dy > xmax - 1)
            {
                h_rst00 = s11.x;
                h_rst01 = s21.x;
            }
            else
            {
                h_rst00 = s11.x * cbufx.x + s12.x * cbufx.y;
                h_rst01 = s21.x * cbufx.x + s22.x * cbufx.y;
            }
            // d0.x = h_rst00 * (1 - fy) + h_rst01 * fy;
            d0.x = h_rst00 * cbufy.x + h_rst01 * cbufy.y;

            // G
            if( dy > xmax - 1)
            {
                h_rst00 = s11.y;
                h_rst01 = s21.y;
            }
            else
            {
                h_rst00 = s11.y * cbufx.x + s12.y * cbufx.y;
                h_rst01 = s21.y * cbufx.x + s22.y * cbufx.y;
            }
            // d0.y = h_rst00 * (1 - fy) + h_rst01 * fy;
            d0.y = h_rst00 * cbufy.x + h_rst01 * cbufy.y;

            // R
            if( dy > xmax - 1)
            {
                h_rst00 = s11.z;
                h_rst01 = s21.z;
            }
            else
            {
                h_rst00 = s11.z * cbufx.x + s12.z * cbufx.y;
                h_rst01 = s21.z * cbufx.x + s22.z * cbufx.y;
            }
            // d0.z = h_rst00 * (1 - fy) + h_rst01 * fy;
            d0.z = h_rst00 * cbufy.x + h_rst01 * cbufy.y;

            //output[(dy*outputWidth + dx) * 3 + 0 ] = (d0.x - mean_value.x)*scale; // B
            //output[(dy*outputWidth + dx) * 3 + 1 ] = (d0.y - mean_value.y)*scale; // G
            //output[(dy*outputWidth + dx) * 3 + 2 ] = (d0.z - mean_value.z)*scale; // R
            //printf("%f %f %f\n", (d0.x - mean_value.x)*scale, (d0.y - mean_value.x)*scale, (d0.z - mean_value.x)*scale);

            //color_type 
            //1:RGB
            //else:BGR
            float3 d_color;
            if (color_type==1)
            {   
                d_color.x=d0.z;
                d_color.y=d0.y;
                d_color.z=d0.x;
            }
            else
            {
                d_color=d0;
            }

            output[0*outputWidth*outputHeight + dy*outputWidth + dx] = (d_color.x - mean_value.x)*scale.x; 
            output[1*outputWidth*outputHeight + dy*outputWidth + dx] = (d_color.y - mean_value.y)*scale.y; 
            output[2*outputWidth*outputHeight + dy*outputWidth + dx] = (d_color.z - mean_value.z)*scale.z; 
        }
    }
}

// cudaPreImageNetMean
hipError_t cudaPreImageScaleMean(unsigned char* input, int inputHeight, int inputWidth, int inputChannels,
                                  float* output, int outputHeight, int outputWidth, float3 scale, const float3& mean_value,const int color_type)
{
    const int inputBytes = inputHeight * inputWidth * inputChannels * sizeof(unsigned char);

    unsigned char *d_input;

    CHECK_COM(hipMalloc<unsigned char>(&d_input, inputBytes));

    CHECK_COM(hipMemcpy(d_input, input, inputBytes, hipMemcpyHostToDevice));
//    printf("*********");
//    std::cout<<("*********")<<std::endl;
//    for(int i=0; i< 100; i++)
//    {
//        std::cout<<int(input[i])<<" ";
//    }
//    std::cout<<int(input[19999])<<" ";
//    std::cout<<int(input[20001])<<" ";
//    std::cout<<std::endl;
    //Specify a reasonable block size
    const dim3 block(16, 16);

    //Calculate grid size to cover the whole image
    const dim3 grid((outputWidth + block.x - 1) / block.x, (outputHeight + block.y - 1) / block.y);

    //Launch the size conversion kernel
    gpuPreImageScaleMean<<<grid, block>>>(d_input, inputWidth, inputHeight, inputChannels,
                                        output, outputWidth, outputHeight, scale, mean_value,color_type);

    CHECK_COM(hipDeviceSynchronize());
    CHECK_COM(hipFree(d_input));

    return CUDA(hipGetLastError());
}

// cudaPreImageNetMean
hipError_t cudaPreImageScaleMeanV2(unsigned char* input, int inputHeight, int inputWidth, int inputChannels,
                                  float* output, int outputHeight, int outputWidth, float3 scale, const float3& mean_value,const int color_type)
{
    //std::cout<<("*********")<<std::endl;
    //const int inputBytes = inputHeight * inputWidth * inputChannels * sizeof(unsigned char);
    //CHECK_COM(hipMalloc<unsigned char>(&d_input, inputBytes));
    //unsigned char *cpu_data;
    //cpu_data=(unsigned char *)malloc(inputBytes);
    //CHECK_COM(hipMemcpy(cpu_data, input, inputBytes, hipMemcpyDeviceToHost));

    //for(int i=0; i< 100; i++)
    //{
    //    std::cout<<int(cpu_data[i])<<" ";
    //}
    //std::cout<<std::endl;
    //Specify a reasonable block size
    const dim3 block(16, 16);

    //Calculate grid size to cover the whole image
    const dim3 grid((outputWidth + block.x - 1) / block.x, (outputHeight + block.y - 1) / block.y);

    //Launch the size conversion kernel
    gpuPreImageScaleMean<<<grid, block>>>(input, inputWidth, inputHeight, inputChannels,
            output, outputWidth, outputHeight, scale, mean_value,color_type);

    CHECK_COM(hipDeviceSynchronize());

    return CUDA(hipGetLastError());
}


// gpuPreImageNetMean
__global__ void gpuPreImageMean(float* output, size_t width, size_t height, float scale, float3 mean_value)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int n = width * height;

    float3 px  = make_float3(output[y * width + x + 0], output[y * width + x + 1], output[y * width + x + 2]);

    float3 bgr = make_float3((px.x - mean_value.x)*scale, (px.y - mean_value.y)*scale, (px.z - mean_value.z)*scale);

    output[n * 0 + y * width + x] = bgr.x;
    output[n * 1 + y * width + x] = bgr.y;
    output[n * 2 + y * width + x] = bgr.z;
}

// cudaPreImageNetMean
hipError_t cudaPreImageMean(float* output, size_t width, size_t height, float scale, float3& mean_value)
{
    if( !output )
        return hipErrorInvalidDevicePointer;

    if( width == 0 || height == 0)
        return hipErrorInvalidValue;

    // launch kernel
    const dim3 blockDim(8, 8);
    const dim3 gridDim(iDivUp(width, blockDim.x), iDivUp(height, blockDim.y));
    gpuPreImageMean<<<gridDim, blockDim>>>(output, width, height, scale, mean_value);

    hipDeviceSynchronize();

    return CUDA(hipGetLastError());
}

// cudaCropImage
__global__ void corpKernel(const unsigned char* input, int inputWidth, int inputChannels, unsigned char* output, int x, int y, int w, int h)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    const int outputWidth = w;

    if (dx >= x && dy >= y && dx < (x+w) && dy < (y+h))
    {
	  if (inputChannels == 3)
	  {
	   output[(outputWidth*(dy-y) + (dx-x))*3 + 0] = input[(inputWidth*dy + dx)*3 + 0];
	   output[(outputWidth*(dy-y) + (dx-x))*3 + 1] = input[(inputWidth*dy + dx)*3 + 1];
	   output[(outputWidth*(dy-y) + (dx-x))*3 + 2] = input[(inputWidth*dy + dx)*3 + 2];
	  }
	  
	  if (inputChannels == 4)
	  {
	   output[(outputWidth*(dy-y) + (dx-x))*4 + 0] = input[(inputWidth*dy + dx)*4 + 0];
	   output[(outputWidth*(dy-y) + (dx-x))*4 + 1] = input[(inputWidth*dy + dx)*4 + 1];
	   output[(outputWidth*(dy-y) + (dx-x))*4 + 2] = input[(inputWidth*dy + dx)*4 + 2];
	   output[(outputWidth*(dy-y) + (dx-x))*4 + 3] = input[(inputWidth*dy + dx)*4 + 3];
	  }
    }
}

hipError_t cudaCropImage(const unsigned char* input, int inputWidth, int inputHeight, int inputChannels,
        unsigned char* output, int x, int y, int w, int h)
{
    if( !output )
        return hipErrorInvalidDevicePointer;

    assert(inputChannels == 3 || inputChannels == 4);  // BGR or BGRA


    assert(x >= 0 && y >= 0 && (x+w) <= inputWidth && (y+h) <= inputHeight);

    // launch kernel
    const dim3 blockDim(8, 8);
    const dim3 gridDim(iDivUp(inputWidth, blockDim.x), iDivUp(inputHeight, blockDim.y));

    corpKernel<<<gridDim, blockDim>>>(input, inputWidth, inputChannels, output, x, y, w, h);

    //if(output == NULL){ 
    //    std::cout<<"OUTPUT is NULL"<<std::endl;
    //}else{
    //    std::cout<<"OUTPUT is NOTNULL"<<std::endl;
    //}

    hipDeviceSynchronize();

    return CUDA(hipGetLastError());
}
#if 0
// cudaCropImage

__global__ void corpKernel(const unsigned char* input, int inputWidth, unsigned char* output, int x1, int y1, int x2, int y2)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    const int outputWidth = x2 - x1;

    if (dx >= x1 && dy >= y1 && dx <= x2 && dy <= y2)
    {
        output[outputWidth*(dy-y1) + (dx-x1) + 0] = input[inputWidth*dy + dx + 0];
        output[outputWidth*(dy-y1) + (dx-x1) + 1] = input[inputWidth*dy + dx + 1];
        output[outputWidth*(dy-y1) + (dx-x1) + 2] = input[inputWidth*dy + dx + 2];
    }
}

hipError_t cudaCropImage(const unsigned char* input, int inputWidth, int inputHeight, int inputChannels,
        unsigned char* output, int x1, int y1, int x2, int y2)
{
    if( !output )
        return hipErrorInvalidDevicePointer;

    assert(inputChannels == 3 || inputChannels == 4);  // BGR or BGRA

    assert( x2 > x1 && y2> y1);

    assert(x1>=0 && y1>=0 && x2<=inputWidth && y2<=inputHeight);

    // launch kernel
    const dim3 blockDim(8, 8);
    const dim3 gridDim(iDivUp(inputWidth, blockDim.x), iDivUp(inputHeight, blockDim.y));

    corpKernel<<<gridDim, blockDim>>>(input, inputWidth, output, x1, y1, x2, y2);

    hipDeviceSynchronize();

    return CUDA(hipGetLastError());
}
#endif
}
